#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#define THREADS_PER_BLOCK 64


__global__ void MatrixMul( float *Md , float *Nd , float *Pd , const int WIDTH )
{
	


int COL = threadIdx.x + blockIdx.x * blockDim.x;
int ROW = threadIdx.y + blockIdx.y * blockDim.y;



    if (ROW < WIDTH && COL < WIDTH) {        
        for (int i = 0; i < WIDTH; i++) {
           Pd[ROW * WIDTH + COL] += Md[ROW * WIDTH + i] * Nd [i * WIDTH + COL];
        }
    }
   
}




int main(int arg0, char *arg1[]) {

  hipDeviceSynchronize();
	
  int WIDTH = atoi(arg1[1]);
  int sqrtThreads = sqrt(THREADS_PER_BLOCK);
  int nBlocks = WIDTH/sqrtThreads;
  
	if (WIDTH % sqrtThreads != 0)
		{ 
			nBlocks++;
		}

  dim3 grid(nBlocks, nBlocks, 1);
  dim3 block(sqrtThreads, sqrtThreads, 1); 


  float *a_h, *b_h, *c_h, *d_h, *a_d, *b_d, *c_d;
  int size;

  hipEvent_t start;
  hipEvent_t stop;
  float elapsed1;

  size = WIDTH * WIDTH * sizeof(float);
  
  a_h = (float*) malloc(size);
  b_h = (float*) malloc(size);
  c_h = (float*) malloc(size);
  d_h = (float*) malloc(size);


	for (int i = 0; i < WIDTH; i++)
	{
		for (int j = 0; j < WIDTH; j++)
		{
			a_h[i * WIDTH + j] = i;
			b_h[i * WIDTH + j] = i;
		}
	}


  hipMalloc((void**)&a_d, size);
  hipMalloc((void**)&b_d, size);
  hipMalloc((void**)&c_d, size);

  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
  hipMemcpy(c_d, c_h, size, hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

 MatrixMul<<<grid, block>>>(a_d, b_d, c_d, WIDTH);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed1, start, stop);

  printf("%f\n", elapsed1/1000);

  hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost); 

  free(a_h);
  free(b_h);
  free(c_h);
  free(d_h);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
