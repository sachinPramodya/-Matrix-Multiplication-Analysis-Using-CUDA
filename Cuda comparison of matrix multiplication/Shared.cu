#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#define THREADS_PER_BLOCK 64

const int TILE_WIDTH = 8;

__global__ void multi(float *a, float *b, float *c, int width) {
  __shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
  __shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float result = 0;
  
	for (int p = 0; p < width/TILE_WIDTH; p++)
	{
		s_a[threadIdx.y][threadIdx.x] = a[row*width + (p*TILE_WIDTH + threadIdx.x)];
		s_b[threadIdx.y][threadIdx.x] = b[(p*TILE_WIDTH + threadIdx.y)*width + col];

		__syncthreads();

		for (int i = 0; i < TILE_WIDTH; i++)
		{
			result += s_a[threadIdx.y][i] * s_b[i][threadIdx.x];
		}

		__syncthreads();
	}

  c[row * width + col] = result;
}



int main(int arg0, char **arg1) {
   hipDeviceSynchronize();
	
  int width = atoi(arg1[1]);

  int sqrtThreads = sqrt(THREADS_PER_BLOCK);
  int nBlocks = width/sqrtThreads;

	if (width % sqrtThreads != 0)
	{ 
		nBlocks++;
	}

  dim3 grid(nBlocks, nBlocks, 1);
  dim3 block(sqrtThreads, sqrtThreads, 1); 


  float *a_h, *b_h, *c_h, *d_h, *a_d, *b_d, *c_d;
  int size;

  hipEvent_t start;
  hipEvent_t stop;
  float elapsed1;

  size = width * width * sizeof(float);
  
  a_h = (float*) malloc(size);
  b_h = (float*) malloc(size);
  c_h = (float*) malloc(size);
  d_h = (float*) malloc(size);


	for (int i = 0; i < width; i++)
	{
		for (int j = 0; j < width; j++)
		{
			a_h[i * width + j] = i;
			b_h[i * width + j] = i;
		}
	}


  hipMalloc((void**)&a_d, size);
  hipMalloc((void**)&b_d, size);
  hipMalloc((void**)&c_d, size);

  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
  hipMemcpy(c_d, c_h, size, hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  multi<<<grid, block>>>(a_d, b_d, c_d, width);
  hipDeviceSynchronize();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed1, start, stop);

  printf("%f\n", elapsed1/1000);

  hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

  free(a_h);
  free(b_h);
  free(c_h);
  free(d_h);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
